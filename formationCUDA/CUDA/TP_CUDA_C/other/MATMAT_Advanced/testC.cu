#include "hip/hip_runtime.h"
#include "test.h"


/**
 * This can be set to true to let cuBlas handle some data transfer.
 * Should not make much of a difference (cuBlas routines use CUDA Runtime routines)
 */
#define USE_CUBLAS_COPY false

//CUDA_SAVE_CALL (and cuBLAS)
#define CUDA_SAFE_CALL(call) { gpuAssert((call), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
inline void gpuAssert(hipblasStatus_t code, const char *file, int line, bool abort=false){
	if(code != HIPBLAS_STATUS_SUCCESS){
		std::cout << "something went wrong in cublas" << "\n";
	}
}

void TestC::printCudaStats(){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	//std::cout << "number of devices: " << nDevices << "\n";
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		
		//std::cout << "Shared memory per block: " << (prop.sharedMemPerBlock / 1024) << " KB\n";
		//std::cout << "canMapHostMemory: " << (prop.canMapHostMemory ? "yes" : "no") << "\n";
	}
}

void TestC::initLibs(REAL *r){
	printCudaStats();
	REAL x[16] = {0, 0, 0, 0, 0, 0, 0, 0, -28, 0, 41, -15, 0, 20, -55.744289, -76.930283};
	REAL y[16] = {0, 0, 0, 0, 0, 0, 0, 0, -54.654705, 0, 23.670898, -4.533447, 0, 79.964569, -65.6026, -4};

	
	REAL expected = .0f, actual = .0f;
    for (int i = 0; i < 16; i++) {
        expected += x[i] * y[i];
    }
	
	handle = new hipblasHandle_t();
	REAL *devPtrA, *devPtrB;
	
    CUDA_SAFE_CALL(hipMalloc((void **)&devPtrA, 16 * sizeof(REAL)));
    CUDA_SAFE_CALL(hipMalloc((void **)&devPtrB, 16 * sizeof(REAL)));
	
	CUDA_SAFE_CALL(hipMemcpy(devPtrA, x, 16*sizeof(REAL), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(devPtrB, y, 16*sizeof(REAL), hipMemcpyHostToDevice));
	
	CUDA_SAFE_CALL(hipblasCreate(handle));
	
	CUDA_SAFE_CALL(hipblasDdot(*handle, 16, devPtrA, 1, devPtrB, 1, &actual));
	
    (*r) = fabs(expected -  actual);
}

void TestC::execTest(const REAL * __restrict__ A, const REAL * __restrict__ B, REAL * __restrict__ C) const {
	hipEvent_t timer_start, timer_stop;
	float time;
    REAL *devPtrA, *devPtrB, *devPtrC;
	
	CUDA_SAFE_CALL(hipEventCreate(&timer_start));
	CUDA_SAFE_CALL(hipEventCreate(&timer_stop));
	
	CUDA_SAFE_CALL(hipEventRecord( timer_start, 0 ));
	
    CUDA_SAFE_CALL(hipMalloc ((void**)&devPtrA, N*N*sizeof(REAL)));
	CUDA_SAFE_CALL(hipMalloc ((void**)&devPtrB, N*N*sizeof(REAL)));
	CUDA_SAFE_CALL(hipMalloc ((void**)&devPtrC, N*N*sizeof(REAL)));
	
	if(USE_CUBLAS_COPY){
		CUDA_SAFE_CALL(hipblasSetMatrix(N, N, sizeof(REAL), A, N, devPtrA, N));
		CUDA_SAFE_CALL(hipblasSetMatrix(N, N, sizeof(REAL), B, N, devPtrB, N));
	}else{
		CUDA_SAFE_CALL(hipMemcpy(devPtrA, A, N*N*sizeof(REAL), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(devPtrB, B, N*N*sizeof(REAL), hipMemcpyHostToDevice));
	}
	
	if(handle == nullptr)
		CUDA_SAFE_CALL(hipblasCreate(handle));
	
	REAL alpha = 1.0f;
	REAL beta = 0.0f;
	CUDA_SAFE_CALL(hipblasDgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, devPtrA, N, devPtrB, N, &beta, devPtrC, N));
	CUDA_SAFE_CALL(hipblasDgeam(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, &alpha, devPtrC, N, &beta, devPtrB, N, devPtrB, N));	//transform from c-major to r-major
	
	if(USE_CUBLAS_COPY){//devPtrB contains the transpose
		//CUDA_SAFE_CALL(hipblasGetMatrix(N,N,sizeof(REAL), devPtrB, N, C, N));	
		CUDA_SAFE_CALL(hipblasGetMatrix(N,N,sizeof(REAL), devPtrB, N, C, N));	
	}else{
		CUDA_SAFE_CALL(hipMemcpy(C, devPtrB, N * N * sizeof(REAL), hipMemcpyDeviceToHost));
	}
	
	CUDA_SAFE_CALL(hipEventRecord( timer_stop, 0 ));
	CUDA_SAFE_CALL(hipEventSynchronize( timer_stop ));

	CUDA_SAFE_CALL(hipEventElapsedTime( &time, timer_start, timer_stop ));
	
	//std::cout << "cuda time: " << time << "\n";
	
	hipFree(devPtrA);
	hipFree(devPtrB);
	hipFree(devPtrC);
	
	hipEventDestroy(timer_start);
	hipEventDestroy(timer_stop);
	
    hipblasDestroy(*handle);
	hipDeviceSynchronize();
}

