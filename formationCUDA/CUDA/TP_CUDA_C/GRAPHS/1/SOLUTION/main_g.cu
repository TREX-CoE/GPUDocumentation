#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <omp.h>

#define CUDA_SAFE_CALL(ans) (ans)
/*
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
*/

__global__ void vec_add(float *a, const float *b, size_t n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(idx < n){
		a[idx] = a[idx] + b[idx];
	}
}


int main(int argc, char **argv){
	int timesteps;
	if(argc == 2)
		timesteps = atoi(argv[1]);
	else 
		timesteps = 100;
	const int N = 2<<19;	//524288, will spin the kernels for a couple of ys 
	float *h_a, *h_b;
	float *d_a, *d_b;
	
	h_a = (float *) malloc(sizeof(float) * N);
	h_b = (float *) malloc(sizeof(float) * N);
	
	CUDA_SAFE_CALL( hipMalloc(&d_a, sizeof(float) * N) );
	CUDA_SAFE_CALL( hipMalloc(&d_b, sizeof(float) * N) );
	
	for(int i = 0; i < N; i++){
		h_a[i] = 2.0f;
		h_b[i] = 4.0f;
	}
	
	CUDA_SAFE_CALL( hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice) );	//from here, we keep the data on the GPU
	
	hipStream_t computeStream;
	CUDA_SAFE_CALL( hipStreamCreate(&computeStream) );

	int blocksize = 256;
	int gridsize = (N + blocksize-1)/blocksize;

	double start = omp_get_wtime();
	
	//setup graph
	bool hasGraph = false;
	hipGraph_t graph;
	hipGraphExec_t instance;
	
	//compute loop
	for(int i = 0; i < timesteps; i++){
		if(!hasGraph){
			CUDA_SAFE_CALL( hipStreamBeginCapture(computeStream) );
			for(int j = 0; j < 20; j++){
				vec_add<<<gridsize, blocksize, 0, computeStream>>>(d_a, d_b, N);
			}
			CUDA_SAFE_CALL( hipStreamEndCapture(computeStream, &graph) );
			CUDA_SAFE_CALL( hipGraphInstantiate(&instance, graph, NULL, NULL, 0) );
			hasGraph = true;//std::cout << "Graph created\n";
		}
		CUDA_SAFE_CALL( hipGraphLaunch(instance, computeStream) );
		
		//lauch a series of small compute kernels
		for(int j = 0; j < 20; j++){
			h_a[0] = h_a[0] + h_b[0];
		}
		CUDA_SAFE_CALL( hipStreamSynchronize(computeStream) );
	}

	double elapsed = omp_get_wtime() - start;

	//test the result
	float test = .0f;
	CUDA_SAFE_CALL( hipMemcpy(&test, d_a, sizeof(float), hipMemcpyDeviceToHost) );
	if(test == h_a[0]){
		std::cout << "Success! elapsed " << elapsed << "\n";
	}else{
		std::cout << "Error! test: " << test << " h_a[0]: " << h_a[0] << "\n";
	}

	return EXIT_SUCCESS;
}