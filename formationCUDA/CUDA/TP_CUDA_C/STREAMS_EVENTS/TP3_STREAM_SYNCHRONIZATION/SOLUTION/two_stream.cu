
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>



double wallclock(){
  struct timeval timer;
  gettimeofday(&timer, NULL);
  double time = timer.tv_sec + timer.tv_usec * 1.0E-6;
  return time;
}



__global__ void myKernel(int size, int *d_i){


  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < size){
    for(int loop=0; loop<10000; loop++)
      d_i[tid] += tid + loop;
  }
}

int main(int argc, char **argv){


  int i,size;
  size = 100000000;

  //allocation and initialization on the host
  int *h_array1;
  int *h_array2;

  hipHostMalloc( (void **) &h_array1, size * sizeof(int), hipHostMallocDefault);
  hipHostMalloc( (void **) &h_array2, size * sizeof(int), hipHostMallocDefault);


  for(i=0;i<size;i++){
    h_array1[i] = 0;
    h_array2[i] = 0;
  }


  hipStream_t stream [2];
  //stream creation
  hipStreamCreate (&stream[0]);
  hipStreamCreate (&stream[1]);


  dim3 threadsPerBlock;
  dim3 numBlocks;
  threadsPerBlock.x = 256;
  numBlocks.x = (size + threadsPerBlock.x -1) / threadsPerBlock.x;


  //allocation on the device
  int *d_array1;
  int *d_array2;
  hipMalloc( (void**) &d_array1, size * sizeof(int));
  hipMalloc( (void**) &d_array2, size * sizeof(int));

  double t0 = wallclock();

  hipMemcpyAsync(d_array1, h_array1, size * sizeof(int), hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(d_array2, h_array2, size * sizeof(int), hipMemcpyHostToDevice, stream[1]);



  myKernel<<< numBlocks, threadsPerBlock, 0, stream[0] >>>(size, d_array1);
  double t1 = wallclock();
  myKernel<<< numBlocks, threadsPerBlock, 0, stream[1] >>>(size, d_array2);



  hipMemcpyAsync(h_array1, d_array1, size * sizeof(int), hipMemcpyDeviceToHost,  stream[0]);
  hipMemcpyAsync(h_array2, d_array2, size * sizeof(int), hipMemcpyDeviceToHost,  stream[1]);

  double t2 = wallclock();
  printf("t1-t0: %lf   t2-t1: %lf   t2-t0: %lf\n",t1-t0,t2-t1,t2-t0);

  hipDeviceSynchronize();
  //stream destruction
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);



  //free device and host memory
  hipFree(d_array1);
  hipFree(d_array2);
  hipHostFree(h_array1);
  hipHostFree(h_array2);

  return 0;
}
