
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void  copy2d(int size_x, int size_y, int *d_in, int *d_out){

  int tx = threadIdx.x + blockIdx.x * blockDim.x;
  int ty = threadIdx.y + blockIdx.y * blockDim.y;
  
  d_out[tx + ty *size_x] = d_in[tx + ty *size_x];

}


int main(int argc, char **argv){

  int size_x = 512;
  int size_y = 512;
  int size_all = size_x * size_y;

  //host allocation
  int* h_in  = (int *) malloc(size_all * sizeof(int));
  int* h_out = (int *) malloc(size_all * sizeof(int));


  //host initialization
  for(int i=0; i<size_all; i++){
    h_in [i] =  i;
    h_out[i] = -1;
  }


  //GPU allocation
  int *d_in;
  int *d_out;
  hipMalloc( (void**) &d_in , size_all * sizeof(int) );
  hipMalloc( (void**) &d_out, size_all * sizeof(int) );


  //copy h_in in d_in and h_out in d_out
  hipMemcpy(d_in , h_in , size_all*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, h_out, size_all*sizeof(int), hipMemcpyHostToDevice);
  
  
  //kernel
  dim3 blocksize;
  dim3 gridsize;

  blocksize.x = 32;
  blocksize.y = 32;
  gridsize.x = (size_x + blocksize.x - 1) / blocksize.x;
  gridsize.y = (size_y + blocksize.y - 1) / blocksize.y;

  copy2d<<<gridsize, blocksize>>>(size_x, size_y, d_in, d_out);

  //copy d_out in h_out
  hipMemcpy(h_out, d_out, size_all*sizeof(int), hipMemcpyDeviceToHost);

  //check resulst
  int success = 1;
  int first_index_error = -1;

  for(int i=0; i<size_all; i++){
    //printf("%d %d\n", h_out[i], h_in[i]);
    if(h_out[i] != h_in[i] ){
      success = 0;
      first_index_error = i;
      break;
    }
  }

  if(success){
    printf("SUCCESS\n");
  }else{
    printf("ERROR: h_out[%d]=%d expected %d\n",first_index_error,h_out[first_index_error], h_in[first_index_error]);
  }


  //free GPU
  hipFree(d_in );
  hipFree(d_out);

  //free host
  free(h_in );
  free(h_out);

  return 0;

}
