#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>


//Complete the kernel
__global__ void myKernel(int size, int offset, float *d_in, float *d_out){

	int tid = ...
		if(tid < size){
			//1D acces + offset
			d_out[...] = d_in[...];
		}

}



int main(int argc, char **argv){

	int offset = 0;

	if(argc == 2){
		int tmp_offset = atoi(argv[1]);

		if(tmp_offset < 0 || tmp_offset > 256){
			printf("Error offset must be > 0 and < 256\n");
			return -1;
		}
		offset = tmp_offset;
	}


	int size;
	// 100 000 256
	size = 100000256;

	//allocation and initialization on the host
	float *h_in, *h_out;

	h_in  = (float *) malloc( size * sizeof(float));
	h_out = (float *) malloc( size * sizeof(float));


	int i;
	for(i=0;i<size;i++){
		h_in[i]  = i;
		h_out[i] = 0;
	}


	dim3 threadsPerBlock;
	dim3 numBlocks;
	threadsPerBlock.x = 256;
	numBlocks.x = (size + threadsPerBlock.x -1) / threadsPerBlock.x;


	//allocation on the device
	float *d_in, *d_out;
	hipMalloc( (void**) &d_in , size * sizeof(float));
	hipMalloc( (void**) &d_out, size * sizeof(float));


	//inialization on the device (copy of the host value)
	hipMemcpy(...);
	hipMemcpy(...);

	//create two events for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//first time measure
	hipEventRecord(start, 0);

	//launch mykernel
	myKernel<<< numBlocks, threadsPerBlock >>>(size, offset, d_in, d_out);

	//second time measure
	hipEventRecord(stop, 0);

	//synchronize and compute the elasped time between the two events
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTimem, start, stop); 

	printf("offset: %d  time: %lf\n",offset, elapsedTime);


	//copy the value of d_i in h_i
	hipMemcpy(h_out, d_out, size * sizeof(float), hipMemcpyDeviceToHost);


	//print a result
	//printf("value: %lf\n", h_out[50]);


	//free device and host memory
	hipFree(d_in );
	hipFree(d_out);
	free(h_in);
	free(h_out);


	return 0;
}
