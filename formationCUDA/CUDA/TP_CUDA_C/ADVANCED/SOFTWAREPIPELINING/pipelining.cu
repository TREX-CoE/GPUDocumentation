#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

//must stay 32x32
#define BLOCK_X 32
#define BLOCK_Y 32

extern "C" __global__ void myKernel_ref(int size_x, int size_y, int size_z, int* in, int* out){

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int idx = tx + blockIdx.x * blockDim.x;
  int idy = ty + blockIdx.y * blockDim.y;
  int idz = 0;

  int temp;

  for(idz = 4; idz < (size_z-4); idz++){

    temp = in[idx + idy *size_x + (idz+0) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+1) * size_x * size_y] - in[idx + idy *size_x + (idz-1) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+2) * size_x * size_y] - in[idx + idy *size_x + (idz-2) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+3) * size_x * size_y] - in[idx + idy *size_x + (idz-3) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+4) * size_x * size_y] - in[idx + idy *size_x + (idz-4) * size_x * size_y];

    if( (idx<size_x) && (idy<size_y) ){
      out[idx + idy *size_x + idz * size_x * size_y] = temp;
    }

  }

}


extern "C" __global__ void myKernel_restrict(int size_x, int size_y, int size_z, const int* __restrict__ in, int* __restrict__ out){

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int idx = tx + blockIdx.x * blockDim.x;
  int idy = ty + blockIdx.y * blockDim.y;
  int idz = 0;

  int temp;
  
  for(idz = 4; idz < (size_z-4); idz++){

    temp = in[idx + idy *size_x + (idz+0) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+1) * size_x * size_y] - in[idx + idy *size_x + (idz-1) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+2) * size_x * size_y] - in[idx + idy *size_x + (idz-2) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+3) * size_x * size_y] - in[idx + idy *size_x + (idz-3) * size_x * size_y]
      +    in[idx + idy *size_x + (idz+4) * size_x * size_y] - in[idx + idy *size_x + (idz-4) * size_x * size_y];

    if( (idx<size_x) && (idy<size_y) ){
      out[idx + idy *size_x + idz * size_x * size_y] = temp; 
    }

  }
  
}


extern "C" __global__ void myKernel_pipeline(int size_x, int size_y, int size_z, const int* __restrict in, int* __restrict__ out){

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int idx = tx + blockIdx.x * blockDim.x;
  int idy = ty + blockIdx.y * blockDim.y;
  int idz = 0;


  //load in register the first memory accesses
  //initalization of the pipeline
  int in_m4 = ...
  int in_m3 = ...
  int in_m2 = ...
  int in_m1 = ...
  int in_cu = ...
  int in_p1 = ...
  int in_p2 = ...
  int in_p3 = ...
  int in_p4 = ...
  

  for(idz = 4; idz < (size_z-4); idz++){

    //pipeline
    in_m4 = ...
    in_m3 = ...
    in_m2 = ...
    in_m1 = ...
    in_cu = ...
    in_p1 = ...
    in_p2 = ...
    in_p3 = ...
    in_p4 = ...
    
    int temp = in_cu
          +    ...
          +    ...
          +    ...
          +    ...

    if( (idx<size_x) && (idy<size_y) ){
      out[idx + idy *size_x + idz * size_x * size_y] = temp; 
    }

  }
  
}




int main(int argc, char **argv){

  int i;

  int size_x = 512;
  int size_y = 512;
  int size_z = 512;

  int size_all = size_x * size_y * size_z;

  int *in  = (int *) malloc(size_all * sizeof(int));
  int *out1 = (int *) malloc(size_all * sizeof(int));
  int *out2 = (int *) malloc(size_all * sizeof(int));
  int *out3 = (int *) malloc(size_all * sizeof(int));

  for(i=0; i<size_all; i++){
    in  [i] = i;
    out1[i] = 0;
    out2[i] = 0;
    out3[i] = 0;
  }


  //CUDA PART
  int* d_in;
  int* d_out1;
  int* d_out2;
  int* d_out3;


  hipMalloc((void **)&d_in  ,size_all*sizeof(int));
  hipMalloc((void **)&d_out1,size_all*sizeof(int));
  hipMalloc((void **)&d_out2,size_all*sizeof(int));
  hipMalloc((void **)&d_out3,size_all*sizeof(int));  

  hipMemcpy(d_in  , in  , size_all*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out1, out1, size_all*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out2, out2, size_all*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out3, out3, size_all*sizeof(int), hipMemcpyHostToDevice);


  dim3 dimBlock;
  dim3 dimGrid;

  dimBlock.x = BLOCK_X;
  dimBlock.y = BLOCK_Y;

  dimGrid.x = (size_x + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (size_y + dimBlock.y - 1) / dimBlock.y;

  //WARMUP
  myKernel_ref     <<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out1);
  myKernel_restrict<<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out2);
  myKernel_pipeline<<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out3);

  //RUN
  myKernel_ref     <<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out1);
  myKernel_restrict<<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out2);
  myKernel_pipeline<<<dimGrid, dimBlock>>>(size_x, size_y, size_z, d_in, d_out3);


  hipMemcpy(out1, d_out1, size_all*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(out2, d_out2, size_all*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(out3, d_out3, size_all*sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_in );
  hipFree(d_out1);
  hipFree(d_out2);
  hipFree(d_out3);

  //CHECK RESULTS
  for(i=0; i<size_all; i++){

    if(out2[i] != out1[i]){
      printf("error out1[%d] != out2[%d] : %d %d\n", i,i,out1[i],out2[i]);
      return -1;
    }
    if(out3[i] != out1[i]){
      printf("error out1[%d] != out3[%d] : %d %d\n", i,i,out1[i],out3[i]);
      return -1;
    }

  }


  free(in );
  free(out1);
  free(out2);
  free(out3);

  return 0;

}
