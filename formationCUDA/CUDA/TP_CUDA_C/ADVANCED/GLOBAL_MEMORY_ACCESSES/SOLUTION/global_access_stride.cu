
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


//Complete the kernel
__global__ void myKernel(int size, int stride, float *d_in, float *d_out){

  int tid = threadIdx.x + blockIdx.x * blockDim.x; 
  if(tid < size){
    //1D acces with stride
    d_out[tid * stride] = d_in[tid * stride];
  }

}


int main(int argc, char **argv){


  int stride = 1;


  if(argc == 2){
    int tmp_stride = atoi(argv[1]);

    if(tmp_stride < 1 || tmp_stride > 32){
      printf("Error stride must be > 0 and < 33\n");
      return -1;
    }
    stride = tmp_stride;
  }


  int size;
  // 100 000 256
  size = 100000256;

  //allocation and initialization on the host
  float *h_in, *h_out;

  h_in  = (float *) malloc( size * sizeof(float));
  h_out = (float *) malloc( size * sizeof(float));


  int i;
  for(i=0;i<size;i++){
    h_in[i]  = i;
    h_out[i] = 0;
  }


  dim3 threadsPerBlock;
  dim3 numBlocks;
  threadsPerBlock.x = 256;
  numBlocks.x = ((size/32) + threadsPerBlock.x -1) / threadsPerBlock.x;


  //allocation on the device
  float *d_in, *d_out;
  hipMalloc( (void**) &d_in , size * sizeof(float));
  hipMalloc( (void**) &d_out, size * sizeof(float));


  //inialization on the device (copy of the host value)
  hipMemcpy(d_in , h_in , size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, h_out, size * sizeof(float), hipMemcpyHostToDevice);

  //create two events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  //first time measure
  hipEventRecord(start, 0);

  //launch mykernel
  myKernel<<< numBlocks, threadsPerBlock >>>(size, stride, d_in, d_out);

  //second time measure
  hipEventRecord(stop, 0);
  
  //synchronize and compute the elasped time between the two events
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("stride: %d  time: %lf\n",stride, elapsedTime);


  //copy the value of d_i in h_i
  hipMemcpy(h_out, d_out, size * sizeof(float), hipMemcpyDeviceToHost);


  //print a result
  //printf("value: %lf\n", h_out[50]);


  //free device and host memory
  hipFree(d_in );
  hipFree(d_out);
  free(h_in);
  free(h_out);


  return 0;
}
