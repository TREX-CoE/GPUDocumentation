
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>



double wallclock(){
  struct timeval timer;
  gettimeofday(&timer, NULL);
  double time = timer.tv_sec + timer.tv_usec * 1.0E-6;
  return time;
}





__global__ void myKernel(int size, int *d_i){


  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < size){
    for(int loop=0; loop<10000; loop++)
      d_i[tid] += tid + loop;
  }
}

int main(int argc, char **argv){


  int i,size;
  size = 100000000;

  //allocation and initialization on the host
  int *h_array1;
  int *h_array2;

  hipHostMalloc( (void **) &h_array1, size * sizeof(int), hipHostMallocDefault);
  hipHostMalloc( (void **) &h_array2, size * sizeof(int), hipHostMallocDefault);


  for(i=0;i<size;i++){
    h_array1[i] = 0;
    h_array2[i] = 0;
  }


  hipStream_t mystream;
  //stream creation
  hipStreamCreateWithFlags(&mystream, hipStreamNonBlocking);


  dim3 threadsPerBlock;
  dim3 numBlocks;
  threadsPerBlock.x = 256;
  numBlocks.x = (size + threadsPerBlock.x -1) / threadsPerBlock.x;


  //allocation on the device
  int *d_array1;
  int *d_array2;
  hipMalloc( (void**) &d_array1, size * sizeof(int));
  hipMalloc( (void**) &d_array2, size * sizeof(int));


  double t0 = wallclock();

  hipMemcpyAsync(d_array1, h_array1, size * sizeof(int), hipMemcpyHostToDevice, 0);
  myKernel<<< numBlocks, threadsPerBlock, 0, 0 >>>(size, d_array1);
  hipMemcpyAsync(h_array1, d_array1, size * sizeof(int), hipMemcpyDeviceToHost, 0);

  double t1  = wallclock();

  hipMemcpyAsync(d_array2, h_array2, size * sizeof(int), hipMemcpyHostToDevice, mystream);
  myKernel<<< numBlocks, threadsPerBlock, 0, mystream >>>(size, d_array2);
  hipMemcpyAsync(h_array2, d_array2, size * sizeof(int), hipMemcpyDeviceToHost, mystream);

  double t2  = wallclock();

  hipDeviceSynchronize();
  //stream destruction
  hipStreamDestroy(mystream);

  double t3  = wallclock();



  //print the result
  printf("value: %d     t1-t0: %lf    t2-t1: %lf  t3-t2: %lf    t3-t0: %lf  \n", h_array1[50], t1-t0,t2-t1,t3-t2,t3-t0);


  //free device and host memory
  hipFree(d_array1);
  hipFree(d_array2);
  hipHostFree(h_array1);
  hipHostFree(h_array2);

  return 0;
}
